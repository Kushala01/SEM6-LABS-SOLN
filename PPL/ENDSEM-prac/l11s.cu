#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void MatMulElementThreadShared(int *a, int *b, int *c, int width, int block_width, int tile_width) {
    __shared__ int MDs[32][32];  
    __shared__ int NDs[32][32]; 
    int m;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int Row = by * tile_width + ty;
    int Col = bx * tile_width + tx;
    int Pvalue = 0;
    for (m = 0; m < width / tile_width; m++) {
        MDs[ty][tx] = a[Row * width + m * tile_width + tx];
        NDs[ty][tx] = b[(m * tile_width + ty) * width + Col];
        __syncthreads();
        for (int k = 0; k < tile_width; k++) {
            Pvalue += MDs[ty][k] * NDs[k][tx];
        }
        __syncthreads();
    }
    c[Row * width + Col] = Pvalue;
}

int main() {
    int width, block_width, tile_width;
    printf("Enter the width of the matrices: ");
    scanf("%d", &width);
    printf("Enter the block width: ");
    scanf("%d", &block_width);
    printf("Enter the tile width: ");
    scanf("%d", &tile_width);

    int *matA, *matB, *matProd;
    int *da, *db, *dc;

    matA = (int*)malloc(sizeof(int) * width * width);
    matB = (int*)malloc(sizeof(int) * width * width);
    matProd = (int*)malloc(sizeof(int) * width * width);

    printf("\n== Enter elements of Matrix A (%dx%d) ==\n", width, width);
    for (int i = 0; i < width * width; i++) {
        scanf("%d", &matA[i]);
    }

    printf("\n== Enter elements of Matrix B (%dx%d) ==\n", width, width);
    for (int i = 0; i < width * width; i++) {
        scanf("%d", &matB[i]);
    }

    hipMalloc((void **)&da, sizeof(int) * width * width);
    hipMalloc((void **)&db, sizeof(int) * width * width);
    hipMalloc((void **)&dc, sizeof(int) * width * width);

    hipMemcpy(da, matA, sizeof(int) * width * width, hipMemcpyHostToDevice);
    hipMemcpy(db, matB, sizeof(int) * width * width, hipMemcpyHostToDevice);

    dim3 grid_conf(width / block_width, width / block_width);
    dim3 block_conf(block_width, block_width);

    MatMulElementThreadShared<<<grid_conf, block_conf>>>(da, db, dc, width, block_width, tile_width);

    hipMemcpy(matProd, dc, sizeof(int) * width * width, hipMemcpyDeviceToHost);

    printf("\n-=Result of Multiplication (%dx%d)=-\n", width, width);
    printf("-----------------------------------\n");
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < width; j++) {
            printf("%6d ", matProd[i * width + j]);
        }
        printf("\n");
    }

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    free(matA);
    free(matB);
    free(matProd);

    return 0;
}