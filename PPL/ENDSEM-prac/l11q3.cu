
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void inclusiveScan(int *input, int *output, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= n) return;

    // Perform parallel reduction (up-sweep phase)
    for (int stride = 1; stride < n; stride *= 2) {
        int index = 2 * stride * (tid + 1) - 1;
        if (index < n) {
            input[index] += input[index - stride];
        }
        __syncthreads();
    }

    // Clear the last element to prepare for down-sweep phase
    if (tid == 0) {
        input[n - 1] = 0;
    }
    __syncthreads();

    // Perform parallel reduction (down-sweep phase)
    for (int stride = n / 2; stride > 0; stride /= 2) {
        int index = 2 * stride * (tid + 1) - 1;
        if (index + stride < n) {
            input[index + stride] += input[index];
        }
        __syncthreads();
    }

    // Write the result back to global memory
    output[tid] = input[tid];
}

int main() {
    const int N = 8;
    const int block_size = 8;

    int input[N] = {3, 1, 7, 0, 4, 1, 6, 3};
    int output[N];

    int *d_input, *d_output;
    hipMalloc(&d_input, N * sizeof(int));
    hipMalloc(&d_output, N * sizeof(int));

    hipMemcpy(d_input, input, N * sizeof(int), hipMemcpyHostToDevice);

    inclusiveScan<<<1, block_size>>>(d_input, d_output, N);

    hipMemcpy(output, d_output, N * sizeof(int), hipMemcpyDeviceToHost);

    printf("Input array:\n");
    for (int i = 0; i < N; i++) {
        printf("%d ", input[i]);
    }
    printf("\n");

    printf("Inclusive Scan (Prefix Sum) Result:\n");
    for (int i = 0; i < N; i++) {
        printf("%d ", output[i]);
    }
    printf("\n");

    hipFree(d_input);
    hipFree(d_output);

    return 0;
}