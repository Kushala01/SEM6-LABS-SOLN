#include<stdio.h>
#include<hip/hip_runtime.h>

#define MAX_SENTENCE_LEN 1024
#define MAX_WORD_LEN 64
#include<string.h>

__global__ void FindMatchingWord(char * d_sen, char * d_word, int * d_count, int sen_len, int word_len){
    int tid=threadIdx.x;
    while(tid<sen_len){
        int i=0;
        while(d_sen[i+tid]==d_word[i] && i<word_len){
            i++;
        }
        if(i==word_len){
            atomicAdd(d_count,1);
            tid+=word_len;
        }else
            tid++;
    }
}

int main(){
    char sen[MAX_SENTENCE_LEN]="what is up what is up";
    char word[MAX_WORD_LEN]="what";
    int word_len,sen_len;
    int count=0;

    char * d_sen, *d_word;
    int *d_count;

    sen_len=strlen(sen);
    word_len=strlen(word);

    hipMalloc((void**)&d_sen,sen_len*sizeof(char));
    hipMalloc((void**)&d_word,word_len*sizeof(char));
    hipMalloc((void**)&d_count,sizeof(int));

    hipMemcpy(d_sen,sen, sen_len*sizeof(char),hipMemcpyHostToDevice);
    hipMemcpy(d_word,word, word_len*sizeof(char),hipMemcpyHostToDevice);

    int blockSize = 1;
    int gridSize = 1;
    FindMatchingWord<<<gridSize,blockSize>>>( d_sen, d_word, d_count,sen_len,word_len);

    hipMemcpy(&count,d_count, sizeof(int),hipMemcpyDeviceToHost);

    printf("\n the word was repeated %d times\n",count);

    hipFree(d_sen);
    hipFree(d_word);
    hipFree(d_count);

    return 0;
}