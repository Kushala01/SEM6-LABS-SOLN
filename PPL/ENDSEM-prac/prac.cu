#include <hip/hip_runtime.h>
#include <stdlib.h>

#include <stdio.h>

__global__ void SPMV_CSR(int *d_data, int *d_col_idx, int *d_row_ptr, int *d_res, int *d_matB, int rowNum) {
    int rowTid = blockIdx.x * blockDim.x + threadIdx.x;
    if (rowTid < rowNum) {
        int start = d_row_ptr[rowTid];
        int end = d_row_ptr[rowTid + 1];
        int sum = 0;
        for (int i = start; i < end; i++) {
            sum += d_data[i] * d_matB[d_col_idx[i]];
        }
        d_res[rowTid] = sum;
    }
}

int main() {
    int r1, c1;
    printf("\nEnter the size of row:\n");
    scanf("%d", &r1);
    printf("\nEnter the size of col:\n");
    scanf("%d", &c1);

    int *matA = (int *)malloc(r1 * c1 * sizeof(int));
    int *matB = (int *)malloc(r1 * sizeof(int));
    int *res = (int *)malloc(r1 * sizeof(int));

    printf("\nEnter the elements of matrix A:\n");
    for (int i = 0; i < r1; i++) {
        for (int j = 0; j < c1; j++) {
            scanf("%d", &matA[i * c1 + j]);
        }
    }

    printf("\nEnter the elements of matrix B:\n");
    for (int i = 0; i < r1; i++) {
        scanf("%d", &matB[i]);
    }

    int nnz = 0;  // Number of non-zero elements in matA
    for (int i = 0; i < r1 * c1; i++) {
        if (matA[i] != 0) {
            nnz++;
        }
    }

    int *data = (int *)malloc(nnz * sizeof(int));
    int *col_idx = (int *)malloc(nnz * sizeof(int));
    int *row_ptr = (int *)malloc((r1 + 1) * sizeof(int));

    int k = 0;
    row_ptr[0] = 0;
    for (int i = 0; i < r1; i++) {
        for (int j = 0; j < c1; j++) {
            if (matA[i * c1 + j] != 0) {
                data[k] = matA[i * c1 + j];
                col_idx[k] = j;
                k++;
            }
        }
        row_ptr[i + 1] = k;
    }

    int *d_data, *d_col_idx, *d_row_ptr, *d_res, *d_matB;
    hipMalloc((void **)&d_data, nnz * sizeof(int));
    hipMalloc((void **)&d_col_idx, nnz * sizeof(int));
    hipMalloc((void **)&d_row_ptr, (r1 + 1) * sizeof(int));
    hipMalloc((void **)&d_res, r1 * sizeof(int));
    hipMalloc((void **)&d_matB, r1 * sizeof(int));

    hipMemcpy(d_data, data, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_idx, col_idx, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row_ptr, row_ptr, (r1 + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_matB, matB, r1 * sizeof(int), hipMemcpyHostToDevice);

    SPMV_CSR<<<(r1 + 255) / 256, 256>>>(d_data, d_col_idx, d_row_ptr, d_res, d_matB, r1);

    hipMemcpy(res, d_res, r1 * sizeof(int), hipMemcpyDeviceToHost);

    printf("\nResultant matrix:\n");
    for (int i = 0; i < r1; i++) {
        printf("%d\n", res[i]);
    }

    hipFree(d_res);
    hipFree(d_data);
    hipFree(d_col_idx);
    hipFree(d_row_ptr);
    hipFree(d_matB);

    free(matA);
    free(matB);
    free(res);
    free(data);
    free(col_idx);
    free(row_ptr);

    return 0;
}