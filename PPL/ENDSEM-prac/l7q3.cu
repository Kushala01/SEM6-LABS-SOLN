#include<stdio.h>
#include<hip/hip_runtime.h>

#define n 10
#define m 5

__global__ void convolution(int *d_inp,int * d_out, int * d_mask){
	int tid=blockDim.x*blockIdx.x+threadIdx.x;
	int start= tid - m/2;
	int ans=0;
	for(int j=0;j<m;j++){
		if(start+j>=0 && start+j<=n){
			ans+=d_inp[start+j]*d_mask[j];
		}
	}d_out[tid]=ans;
}

int main(){
	int inp[n],out[n],mask[m];
	int *d_inp,*d_out, *d_mask;
	printf("\nenter %d digits for input array\n",n);
	for(int i=0;i<n;i++ ){
	scanf("%d",&inp[i]);
	}printf("\nenter %d digits for input array\n",m);
	for(int i=0;i<m;i++ ){
	scanf("%d",&mask[i]);
	}
	hipMalloc((void**)&d_inp,n*sizeof(int));
	hipMalloc((void**)&d_out,n*sizeof(int));
	hipMalloc((void**)&d_mask,m*sizeof(int));

	hipMemcpy(d_inp,inp,n*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_mask,mask,m*sizeof(int),hipMemcpyHostToDevice);

	convolution<<<1,n>>>(d_inp,d_out,d_mask);

	hipMemcpy(out,d_out,n*sizeof(int),hipMemcpyDeviceToHost);

	printf("\n the resultant is \n");
	for(int i=0;i<n;i++)
		printf("%d\n",out[i]);

	hipFree(d_out);
	hipFree(d_inp);
	hipFree(d_mask);
	return 0;
}