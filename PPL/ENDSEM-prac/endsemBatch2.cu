#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include<hip/hip_runtime.h>


__device__ int position(char* str, int* row, int id) {
    int pos = 0;
    for (int i = 0; i < row[id]; i++) {
        if (str[id] == '+')
            pos += row[id] + 1;
        else
            pos += row[id] + 2;
    }
    return pos;
}

__global__ void sparse(char* str, char* res, int* row, int* col, int len) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id > len) return;
        int pos = position(str, row, id);
        if (str[id] == '+') {
            for (int i = pos; i < pos + row[id] + 1; i++)
                res[i] = str[id];
        } else {
            for (int i = pos; i < pos + row[id] + 2; i++)
                res[i] = str[id];
        }
}

int main() {
    char str[100], res[100];

    printf("Enter string: ");
    scanf("%s", str);
    int len = strlen(str);

    int row[len], col[len];

    printf("Enter row mat: ");
    for (int i = 0; i < len; i++)
        scanf("%d", &row[i]);

    printf("Enter col mat: ");
    for (int i = 0; i < len; i++)
        scanf("%d", &col[i]);

    int* d_row, *d_col;
    char* d_str, *d_res;

    hipMalloc((void**)&d_str, len * sizeof(char));
    hipMalloc((void**)&d_res, len * len * sizeof(char)); // Assuming maximum possible length for result
    hipMalloc((void**)&d_row, len * sizeof(int));
    hipMalloc((void**)&d_col, len * sizeof(int));

    hipMemcpy(d_str, str, len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_res, res, len * len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_row, row, len * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col, col, len * sizeof(int), hipMemcpyHostToDevice);

    int numOfThreads = 256;
    int numOfBlocks = (len + numOfThreads - 1) / numOfThreads; // Adjusted block count

    sparse<<<numOfBlocks, numOfThreads>>>(d_str, d_res, d_row, d_col, len);

    hipMemcpy(res, d_res, len * len * sizeof(char), hipMemcpyDeviceToHost);

    printf("Result: %s\n", res);

    hipFree(d_str);
    hipFree(d_res);
    hipFree(d_row);
    hipFree(d_col);

    return 0;
}