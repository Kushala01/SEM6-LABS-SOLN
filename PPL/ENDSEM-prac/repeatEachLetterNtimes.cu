#include <stdio.h>
#include <hip/hip_runtime.h>


#define MAX_LENGTH 256  // Maximum length of input and output strings

__global__ void repeatStringKernel(const char* input, char* output, int length) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int index = tid * 3;  // Each character in input is repeated three times

    if (tid < length) {
        for (int i = 0; i < 3; i++) {
            output[index + i] = input[tid];  // Repeat the character three times
        }
    }
}

int main() {
    char input[MAX_LENGTH];
    char output[MAX_LENGTH * 3];  // Output string can be up to three times the input length
    char *d_input, *d_output;
    int length;

    printf("Enter a string: ");
    scanf("%s", input);

    length = strlen(input);

    hipMalloc((void**)&d_input, length * sizeof(char));
    hipMalloc((void**)&d_output, length * 3 * sizeof(char));  // Allocate space for output

    hipMemcpy(d_input, input, length * sizeof(char), hipMemcpyHostToDevice);

    int blockSize = 256;  // Threads per block
    int gridSize = (length + blockSize - 1) / blockSize;  // Calculate grid size

    repeatStringKernel<<<gridSize, blockSize>>>(d_input, d_output, length);

    hipMemcpy(output, d_output, length * 3 * sizeof(char), hipMemcpyDeviceToHost);

    printf("Input string: %s\n", input);
    printf("Output string: %s\n", output);

    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
