// transpose of matrix

#include "hip/hip_runtime.h"


#include<stdio.h>
#include<stdlib.h>
#include<string.h>

#define N 1024

// row-wise
__global__ void adda(int *a, int* b, int *t, int ncol) {
    int ridx = threadIdx.x;

    for(int i=0; i<ncol; i++)
        t[ridx*ncol+i] = a[ridx*ncol+i] + b[ridx*ncol+i];
  
}

// column-wise
__global__ void addb(int *a, int* b, int *t, int nrow) {
    int cidx = threadIdx.x;
    int ncol = blockDim.x;

    for(int i=0; i<nrow; i++)
        t[cidx+ncol*i] = a[cidx+ncol*i] + b[cidx+ncol*i];
  
}

// element-wise
__global__ void addc(int *a, int* b, int *t) {
    int eidx = threadIdx.x;
    t[eidx] = a[eidx] + b[eidx];
}

void display(int r, int c, int *t)
{
    int i, j;
    for(i=0;i<r;i++)
    {
        for(j=0;j<c;j++)
        {
            printf("%d\t",t[i*c+j]);
            t[i*c+j] = 0;  // CLEARING THE RESULTANT MATRIX SIMULTANEOUSLY
        }   
        printf("\n");
    }
}

int main() 
{
    int *a, *b, *t;
    int r, c, i;
    int *da, *db, *dt;

    printf("Enter number of rows and columns: ");
    scanf("%d %d", &r, &c);

    int size = sizeof(int)*r*c;

    a = (int*)malloc(size);
    b = (int*)malloc(size);
    t = (int*)malloc(size);

    printf("Enter the elements of the matrix A: \n");
    for(i=0;i<r*c;i++)
        scanf("%d", &a[i]);

    printf("\nEnter the elements of the matrix B: \n");
    for(i=0;i<r*c;i++)
        scanf("%d", &b[i]);

    hipMalloc((void**)&da, size);
    hipMalloc((void**)&db, size);
    hipMalloc((void**)&dt, size);
    hipMemcpy(da, a, size, hipMemcpyHostToDevice);
    hipMemcpy(db, b, size, hipMemcpyHostToDevice);
    hipMemcpy(dt, t, size, hipMemcpyHostToDevice);

    // ROW-WISE
    adda<<<1,r>>>(da, db, dt, c);

    hipMemcpy(t, dt, size, hipMemcpyDeviceToHost);
    printf("\nA) Row-wise summation:\n");
    display(r,c,t);
    hipFree(dt);

    hipMalloc((void**)&dt, size);
    hipMemcpy(dt, t, size, hipMemcpyHostToDevice);

    // COLUMN-WISE
    addb<<<1,c>>>(da, db, dt, r);
    
    hipMemcpy(t, dt, size, hipMemcpyDeviceToHost);
    printf("\nB) Column-wise summation:\n");
    display(r,c,t);
    hipFree(dt);

    hipMalloc((void**)&dt, size);
    hipMemcpy(dt, t, size, hipMemcpyHostToDevice);
    
    // ELEMENT-WISE
    addc<<<1,r*c>>>(da, db, dt);
    
    hipMemcpy(t, dt, size, hipMemcpyDeviceToHost);
    printf("\nC) Element-wise summation:\n");
    display(r,c,t);
    hipFree(dt);

    hipFree(da);
    hipFree(db);
    
    return 0;
}
