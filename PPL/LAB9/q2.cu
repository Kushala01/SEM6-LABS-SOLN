#include "hip/hip_runtime.h"
// transpose of matrix

#include "hip/hip_runtime.h"
#include ""

#include<stdio.h>
#include<stdlib.h>
#include<string.h>

#define N 1024

// row-wise - NOT DONE
__global__ void multa(int *a, int* b, int *t, int n, int p) {
    int ridx = threadIdx.x;
    int m = blockDim.x;

    for(int j=0; j<n; j++)
        for(int k=0; k<p; k++)
            t[ridx*ncol+i] = a[ridx*ncol+i] + b[ridx*ncol+i];
  
}

// // column-wise
// __global__ void multb(int *a, int* b, int *t, int nrow) {
//     int cidx = threadIdx.x;
//     int ncol = blockDim.x;

//     for(int i=0; i<nrow; i++)
//         t[cidx+ncol*i] = a[cidx+ncol*i] + b[cidx+ncol*i];
  
// }

// // element-wise
// __global__ void multc(int *a, int* b, int *t) {
//     int eidx = threadIdx.x;
//     t[eidx] = a[eidx] + b[eidx];
// }

void display(int r, int c, int *t)
{
    int i, j;
    for(i=0;i<r;i++)
    {
        for(j=0;j<c;j++)
        {
            printf("%d\t",t[i*c+j]);
            t[i*c+j] = 0;  // CLEARING THE RESULTANT MATRIX SIMULTANEOUSLY
        }   
        printf("\n");
    }
}

int main() 
{
    int *a, *b, *t;
    int m, n, p, i;
    int *da, *db, *dt;

    printf("Enter number of rows and columns of A: ");
    scanf("%d %d", &m, &p);
    printf("Enter number of columns of B: ");
    scanf("%d", &n);

    int sizea = sizeof(int)*m*p;
    int sizeb = sizeof(int)*p*n;
    int sizet = sizeof(int)*m*n;

    a = (int*)malloc(sizea);
    b = (int*)malloc(sizeb);
    t = (int*)malloc(sizet);

    printf("Enter %d elements of the matrix A: \n",m*p);
    for(i=0;i<m*p;i++)
        scanf("%d", &a[i]);

    printf("\nEnter %d elements of the matrix B: \n",p*n);
    for(i=0;i<p*n;i++)
        scanf("%d", &b[i]);

    hipMalloc((void**)&da, sizea);
    hipMalloc((void**)&db, sizeb);
    hipMalloc((void**)&dt, sizet);
    hipMemcpy(da, a, sizea, hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeb, hipMemcpyHostToDevice);
    hipMemcpy(dt, t, sizet, hipMemcpyHostToDevice);

    // ROW-WISE
    multa<<<1,m>>>(da, db, dt, n, p); 

    hipMemcpy(t, dt, sizet, hipMemcpyDeviceToHost);
    printf("\nA) Row-wise multiplication:\n");
    display(m,n,t);
    hipFree(dt);

    // hipMalloc((void**)&dt, size);
    // hipMemcpy(dt, t, size, hipMemcpyHostToDevice);

    // // COLUMN-WISE
    // multb<<<1,c>>>(da, db, dt, r);
    
    // hipMemcpy(t, dt, size, hipMemcpyDeviceToHost);
    // printf("\nB) Column-wise multiplication:\n");
    // display(r,c,t);
    // hipFree(dt);

    // hipMalloc((void**)&dt, size);
    // hipMemcpy(dt, t, size, hipMemcpyHostToDevice);
    
    // // ELEMENT-WISE
    // multc<<<1,r*c>>>(da, db, dt);
    
    // hipMemcpy(t, dt, size, hipMemcpyDeviceToHost);
    // printf("\nC) Element-wise multiplication:\n");
    // display(r,c,t);
    // hipFree(dt);

    hipFree(da);
    hipFree(db);
    
    return 0;
}
