// transpose of matrix

#include "hip/hip_runtime.h"


#include<stdio.h>
#include<stdlib.h>
#include<string.h>

#define N 1024

__global__ void transpose(int *a, int *t) {
    int n = threadIdx.x;
    int m = blockIdx.x;
    int size = blockDim.x;
    int sizel = gridDim.x;

    t[n*sizel+m] = a[m*size+n];
  
}

int main() 
{
    int *a, *t, m, n, i, j;
    int *da, *dt;

    printf("Enter number of rows and columns: ");
    scanf("%d %d", &m, &n);

    int size = sizeof(int)*m*n;

    a = (int*)malloc(m*n*sizeof(int));
    t = (int*)malloc(m*n*sizeof(int));

    printf("Enter the elements of the matrix: \n");
    for(i=0;i<m*n;i++)
        scanf("%d", &a[i]);

    hipMalloc((void**)&da, size);
    hipMalloc((void**)&dt, size);

    hipMemcpy(da, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dt, t, size, hipMemcpyHostToDevice);

    transpose<<<m,n>>>(da, dt);

    hipMemcpy(t, dt, size, hipMemcpyDeviceToHost);

    printf("Result Vector is:\n");

    for(i=0;i<n;i++)
    {
        for(j=0;j<m;j++)
            printf("%d\t",t[i*m+j]);
        printf("\n");
    }


    hipFree(da);
    hipFree(dt);

    return 0;
}
