#include "hip/hip_runtime.h"


#include<stdio.h>
#include<stdlib.h>
#include<string.h>

#define N 1024

__global__ void countOccurrences(char* str, char* w, int* result, int length, int wn) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < length) {
        if (tid + wn <= length) {
            bool match = true;
            for (int i = 0; i < wn; i++) {
                if (str[tid + i] != w[i]) {
                    match = false;
                    break;
                }
            }

            if (match) {
                atomicAdd(result, 1);
            }
        }
    }
}

int main() {
    char a[N];
    char w[N];

    printf("Enter a string: ");
    scanf("%[^\n]s", a);
    printf("Enter a word: ");
    scanf("%s", w);

    int an = strlen(a);
    int wn = strlen(w);

    char* da;
    char* dw;
    int* dres;

    hipMalloc((void**)&da, an * sizeof(char));
    hipMalloc((void**)&dw, wn * sizeof(char));
    hipMalloc((void**)&dres, sizeof(int));

    hipMemcpy(da, a, an * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(dw, w, wn * sizeof(char), hipMemcpyHostToDevice);
    hipMemset(dres, 0, sizeof(int));

    int block_size = 32;
    int num_blocks = (an + block_size - 1) / block_size;

    countOccurrences<<<num_blocks, block_size>>>(da, dw, dres, an, wn);

    int result;
    hipMemcpy(&result, dres, sizeof(int), hipMemcpyDeviceToHost);

    printf("Occurrences of the '%s' is %d\n",w,result);

    hipFree(da);
    hipFree(dw);
    hipFree(dres);

    return 0;
}
