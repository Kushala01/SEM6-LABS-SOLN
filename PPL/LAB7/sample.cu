#include "hip/hip_runtime.h"


#include<stdio.h>
#include<stdlib.h>
#include<string.h>

#define N 1024

__global__ void CUDACount(char *a, unsigned int *dc)
{
    int i = threadIdx.x;
    if(a[i]=='a')
        atomicAdd(dc, 1);
}

int main()
{
    char a[N];
    char *da;
    unsigned int *dc;
    int res;
    printf("Enter a string: ");
    scanf("%s", a);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    hipMalloc((void**)&da, strlen(a)*sizeof(char));
    hipMalloc((void**)&dc, sizeof(unsigned int));

    hipMemcpy(da, a, strlen(a)*sizeof(char), hipMemcpyHostToDevice);

    hipError_t err = hipGetLastError();
    if(err!=hipSuccess)
        printf("CUDA Error 1 : %s\n", hipGetErrorString(err));
    
    CUDACount<<<1,strlen(a)>>>(da, dc);
    err = hipGetLastError();
    if(err!=hipSuccess)
        printf("CUDA Error 2 : %s\n", hipGetErrorString(err));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float et;
    hipEventElapsedTime(&et, start, stop);

    hipMemcpy(&res, dc, sizeof(unsigned int), hipMemcpyDeviceToHost);

    printf("Total occurences of a: %d", res);
    printf("\nTotal time taken: %f", et);

    hipFree(da);
    hipFree(dc);
    printf("\n");
    getchar();

    return 0;

}
