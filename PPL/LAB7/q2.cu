#include "hip/hip_runtime.h"


#include<stdio.h>
#include<stdlib.h>
#include<string.h>

#define N 1024

__global__ void repeatString(char *s, char *rs, int len) {
    int i = threadIdx.x;
    int x = len;
    int sum = 0;

    if (i < len) 
    {
        for(int j=0; j<i; j++,x--)
            sum += x;
        for(int j=0; j<len-i; j++)
            rs[sum++] = s[j];
    }
}

int main() {
    char a[N];

    printf("Enter a string: ");
    scanf("%s", a);

    int an = strlen(a);
    int x = an*(an+1)/2;

    char* da;
    char* dres;

    hipMalloc((void**)&da, an * sizeof(char));
    hipMalloc((void**)&dres, x * sizeof(char));

    hipMemcpy(da, a, an * sizeof(char), hipMemcpyHostToDevice);
    hipMemset(dres, 0, sizeof(int));
        
    repeatString<<<1, an>>>(da, dres, an);
    
    char rs[x];
    hipMemcpy(rs, dres, sizeof(char) * x, hipMemcpyDeviceToHost);
    rs[x]='\0';

    printf("Result: %s\n",rs);

    hipFree(da);
    hipFree(dres);

    return 0;
}





    

    
    
   

    
